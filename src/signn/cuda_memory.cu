#include "common.cuh"

void *AllocDataSpace(size_t nbytes) {
    void *ret = nullptr;
    CUDA_CALL(hipMalloc(&ret, nbytes));
    return ret;
}

void FreeDataSpace(void *ret) {
  CUDA_CALL(hipFree(ret));
}
