#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include "cuda_mapping.cuh"

inline int RoundUpDiv(int target, int unit) {
  return (target + unit - 1) / unit;
}



template <typename IdType,size_t BLOCK_SIZE, size_t TILE_SIZE>
__device__ void map_node_ids(const IdType *const global,
                             IdType *const new_global, const size_t num_input,
                             const DeviceOrderedHashTable<IdType> &table) {
  assert(BLOCK_SIZE == blockDim.x);

  using Bucket = typename OrderedHashTable<IdType>::Mapping;

  const size_t block_start = TILE_SIZE * blockIdx.x;
  const size_t block_end = min(TILE_SIZE * (blockIdx.x + 1), num_input);

  for (size_t idx = threadIdx.x + block_start; idx < block_end;
       idx += BLOCK_SIZE) {
    const Bucket &bucket = *table.Search(global[idx]);
    new_global[idx] = bucket.local;
  }
}

template <typename IdType,size_t BLOCK_SIZE, size_t TILE_SIZE>
__global__ void map_edge_ids(const IdType *const global_src,
                             IdType *const new_global_src,
                             const IdType *const global_dst,
                             IdType *const new_global_dst,
                             const size_t num_edges,
                             DeviceOrderedHashTable<IdType> table) {
  assert(BLOCK_SIZE == blockDim.x);
  assert(2 == gridDim.y);

  if (blockIdx.y == 0) {
    map_node_ids<IdType,BLOCK_SIZE, TILE_SIZE>(global_src, new_global_src, num_edges,
                                        table);
  } else {
    map_node_ids<IdType,BLOCK_SIZE, TILE_SIZE>(global_dst, new_global_dst, num_edges,
                                        table);
  }
}

template <typename IdType>
void GPUMapEdges( IdType * global_src, IdType * new_global_src,
                  IdType * global_dst, IdType * new_global_dst,
                  size_t num_edges, DeviceOrderedHashTable<IdType> table
                ) {
  const int slice = 1024;
  const int blockSize = 256;
  const size_t num_tiles = RoundUpDiv(num_edges,slice);
  const dim3 grid(num_tiles, 2);
  const dim3 block(blockSize);

  map_edge_ids<IdType,blockSize, slice>
      <<<grid, block>>>(global_src, new_global_src, global_dst,
                                      new_global_dst, num_edges, table);
}

template void GPUMapEdges<int32_t>(
    int32_t * global_src, int32_t * new_global_src,
    int32_t * global_dst, int32_t * new_global_dst,
    size_t num_edges, DeviceOrderedHashTable<int32_t> table
);

template void GPUMapEdges<int64_t>(
    int64_t * global_src, int64_t * new_global_src,
    int64_t * global_dst, int64_t * new_global_dst,
    size_t num_edges, DeviceOrderedHashTable<int64_t> table
);
