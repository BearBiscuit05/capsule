#include "common.cuh"
#include "cuda_hashtable.cuh"
#include "cuda_mapping.cuh"
#include "signn.h"

#define NUM 64

int main(){
  // OrderedHashTable<int64_t> table(80);
  // std::vector<int64_t> myVector = {2, 3, 4, 2, 7, 8};
  // myVector.resize(NUM,0);
  // std::vector<int64_t> myUnique(NUM,0);
  
  // int64_t inputNUM = NUM;
  // int64_t unique = 0;
  // table.FillWithDuplicates(myVector.data(),inputNUM,myUnique.data(),&unique);
  // std::cout << "uniqueNUM :"<< unique << std::endl;
  std::vector<int> edges(100,0);
  for (int i = 0 ; i < 100 ; i++) {
    edges[i] = i;
  }
  std::vector<int> bound= {0,2,10,20,20,21,30,40,40,50,50,60,60,70,70,80,80,90,90,100};
  std::vector<int> seed={0,1,2,3};
  int64_t seed_num=4;
  std::vector<int> fanouts={5};
  int64_t fanoutNUM=1;
  int sampledNUM = fanouts[0] * seed_num;
  std::vector<int> outSrcNodes(sampledNUM,0);
  std::vector<int> outDstNodes(sampledNUM,0);
  std::vector<int> outList(2,0);
  std::vector<int> outrawnodesid(sampledNUM,0);
  int64_t outnodesNUM = 0;

  int *dev_edges;int *dev_bound;int *dev_seed;
  int *dev_fanouts;int *dev_outSrcNodes;int *dev_outDstNodes;int *dev_outList;int *dev_outrawnodesid;
  printf("main in... \n");
  CUDA_CALL(hipMalloc(&dev_edges, sizeof(int)*100));
  CUDA_CALL(hipMalloc(&dev_bound, sizeof(int)*20));
  CUDA_CALL(hipMalloc(&dev_seed, sizeof(int)*seed_num));
  CUDA_CALL(hipMalloc(&dev_fanouts, sizeof(int)));
  CUDA_CALL(hipMalloc(&dev_outSrcNodes, sizeof(int)*sampledNUM));
  CUDA_CALL(hipMalloc(&dev_outDstNodes, sizeof(int)*sampledNUM));
  CUDA_CALL(hipMalloc(&dev_outList, sizeof(int)*2));
  CUDA_CALL(hipMalloc(&dev_outrawnodesid, sizeof(int)*sampledNUM));

	CUDA_CALL(hipMemcpy(dev_edges, edges.data(), sizeof(int)*100, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_bound, bound.data(), sizeof(int)*20, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_seed, seed.data(), sizeof(int)*seed_num, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_fanouts, fanouts.data(), sizeof(int), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_outSrcNodes, outSrcNodes.data(), sizeof(int)*sampledNUM, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_outDstNodes, outDstNodes.data(), sizeof(int)*sampledNUM, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_outList, outList.data(), sizeof(int)*2, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_outrawnodesid, outrawnodesid.data(), sizeof(int)*sampledNUM, hipMemcpyHostToDevice));
  printf("get in... \n");
  mutiLayersSample(
    dev_edges,dev_bound,
    dev_seed,seed_num,fanouts.data(),fanoutNUM,
    dev_outSrcNodes,dev_outDstNodes,dev_outList,
    dev_outrawnodesid,outnodesNUM);
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(edges.data(),dev_edges, sizeof(int)*100, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(bound.data(),dev_bound,  sizeof(int)*20, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(seed.data(),dev_seed,  sizeof(int)*seed_num, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(fanouts.data(),dev_fanouts, sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(outSrcNodes.data(),dev_outSrcNodes, sizeof(int)*sampledNUM, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(outDstNodes.data(),dev_outDstNodes,  sizeof(int)*sampledNUM, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(outList.data(), dev_outList, sizeof(int)*2, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(outrawnodesid.data(), dev_outrawnodesid,sizeof(int)*sampledNUM, hipMemcpyDeviceToHost));
  std::cout << std::endl;

  for (int i = 0 ; i  < sampledNUM ; i++) {
    std::cout << outSrcNodes[i] << " ";
  }
  std::cout << std::endl;
  for (int i = 0 ; i  < sampledNUM ; i++) {
    std::cout << outDstNodes[i] << " ";
  }
  std::cout << std::endl;
  return 0;
}