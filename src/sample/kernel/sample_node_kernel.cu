
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void sample_full_kernel(
                            int* outputSRC,
                            int* outputDST,             //output
                            const int* graphEdge,
                            const int* boundList,
                            const int* trainNode,
                            int nodeNUM) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * 25;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        for (int l = 0 ; l < (idxEnd - idxStart) ; l++) {
            outputSRC[writeIdx] = graphEdge[idxStart + l];
            outputDST[writeIdx++] = id;
        }

    }    
}

__global__ void sample1Hop(
                        int* outputSRC1,
                        int* outputDST1, 
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int nodeNUM
                            ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int blockSize = sampleNUM1;
    int seed = nodeNUM * sampleNUM1 * idx;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * blockSize;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        int neirNUM = idxEnd - idxStart;
        for (int l = 0 ; l < neirNUM ; l++) {
            hiprandState state;
            hiprand_init(seed, idx, 0, &state);  
            int random_value = 0;
            random_value = hiprand(&state) % neirNUM;
            outputSRC1[writeIdx] = graphEdge[idxStart + random_value];
            outputDST1[writeIdx++] = id;
        }
        for (int l = neirNUM; l < sampleNUM1 ; l++) {
            outputSRC1[writeIdx] = 0;
            outputDST1[writeIdx++] = id;
        }
    }

}


__global__ void sample2Hop(
                        int* outputSRC1,
                        int* outputDST1,
                        int* outputSRC2,
                        int* outputDST2,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int sampleNUM2,
                        int nodeNUM
                            ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int seed = nodeNUM * sampleNUM2 * idx;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * sampleNUM1;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        int neirNUM = idxEnd - idxStart;
        for (int l = 0 ; l < neirNUM && l < sampleNUM1 ; l++) {
            hiprandState state;
            hiprand_init(seed, idx, 0, &state); 
            int random_value = 0;
            random_value = hiprand(&state) % neirNUM;
            printf("rand value %d\n",random_value);
            outputSRC1[writeIdx] = graphEdge[idxStart + random_value];
            outputDST1[writeIdx++] = id;
            printf("outsrc %d\n",outputSRC1[writeIdx-1]);
        }
        for (int l = neirNUM; l < sampleNUM1 ; l++) {
            outputSRC1[writeIdx] = -1;
            outputDST1[writeIdx++] = id;
        }

        // hop-2
        for (int l1 = 0 ; l1 < sampleNUM1 ; l1++) {
            // 二层采样id
            int l2_id = outputSRC1[i * sampleNUM1 + l1];
            if (l2_id > 0) {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                int l2_idStart = boundList[l2_id];
                int l2_idEnd = boundList[l2_id+1];
                int l2_neirNUM = l2_idEnd - l2_idStart;
                for (int l = 0 ; l < l2_neirNUM && l < sampleNUM2 ; l++) {
                    hiprandState state;
                    hiprand_init(seed*l1, idx, 0, &state); 
                    int random_value = 0;
                    random_value = hiprand(&state) % l2_neirNUM;
                    outputSRC2[l2_writeIdx] = graphEdge[l2_idStart + random_value];
                    outputDST2[l2_writeIdx++] = l2_id;
                }
                for (int l = l2_neirNUM; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            } else {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                for (int l = 0 ; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            }
            
        }
    }
}

__global__ void sample3Hop(
                        int* outputSRC1,int* outputDST1,
                        int* outputSRC2,int* outputDST2,
                        int* outputSRC3,int* outputDST3,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,int sampleNUM2,int sampleNUM3,
                        int nodeNUM
                            ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int random_value = 0;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * sampleNUM1;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        int neirNUM = idxEnd - idxStart;
        for (int l = 0 ; l < neirNUM && l < sampleNUM1 ; l++) {
            int seed = nodeNUM * sampleNUM1 * idx;
            hiprandState state;
            hiprand_init(seed, idx, 0, &state); 
            random_value = hiprand(&state) % neirNUM;  // 生成0到99之间的随机整数
            outputSRC1[writeIdx] = graphEdge[idxStart + random_value];
            outputDST1[writeIdx++] = id;
        }
        for (int l = neirNUM; l < sampleNUM1 ; l++) {
            outputSRC1[writeIdx] = -1;
            outputDST1[writeIdx++] = id;
        }

        // hop-2
        for (int l1 = 0 ; l1 < sampleNUM1 ; l1++) {
            // 二层采样id
            int l2_id = outputSRC1[i * sampleNUM1 + l1];
            if (l2_id > 0) {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                int l2_idStart = boundList[l2_id];
                int l2_idEnd = boundList[l2_id+1];
                int l2_neirNUM = l2_idEnd - l2_idStart;
                for (int l = 0 ; l < l2_neirNUM && l < sampleNUM2 ; l++) {
                    int seed = nodeNUM * sampleNUM2 * idx;
                    hiprandState state;
                    hiprand_init(seed, idx, 0, &state); 
                    random_value = hiprand(&state) % l2_neirNUM; 
                    outputSRC2[l2_writeIdx] = graphEdge[l2_idStart + random_value];
                    outputDST2[l2_writeIdx++] = l2_id;
                }
                for (int l = l2_neirNUM; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            } else {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                for (int l = 0 ; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            }
            
        }

        for (int l2 = 0 ; l2 < sampleNUM2 ; l2++) {
            int l3_id = outputSRC2[i * sampleNUM2 + l2];
            if (l3_id > 0) {
                int l3_writeIdx = i*sampleNUM2*sampleNUM3 + l2*sampleNUM3;
                int l3_idStart = boundList[l3_id];
                int l3_idEnd = boundList[l3_id+1];
                int l3_neirNUM = l3_idEnd - l3_idStart;
                for (int l = 0 ; l < l3_neirNUM && l < sampleNUM3 ; l++) {
                    int seed = nodeNUM * sampleNUM3 * idx;
                    hiprandState state;
                    hiprand_init(seed, idx, 0, &state); 
                    random_value = hiprand(&state) % l3_neirNUM; 
                    outputSRC3[l3_writeIdx] = graphEdge[l3_idStart + random_value];
                    outputDST3[l3_writeIdx++] = l3_id;
                }
                for (int l = l3_neirNUM; l < sampleNUM3 ; l++) {
                    outputSRC3[l3_writeIdx] = -1;
                    outputDST3[l3_writeIdx++] = l3_id;
                }
            } else {
                int l3_writeIdx = i*sampleNUM2*sampleNUM3 + l2*sampleNUM3;
                for (int l = 0 ; l < sampleNUM2 ; l++) {
                    outputSRC3[l3_writeIdx] = -1;
                    outputDST3[l3_writeIdx++] = l3_id;
                }
            }
        }
    }
}

__global__ void generate_random_integers(int* result, unsigned int seed)
{
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, thread_id, 0, &state);

    int random_value = hiprand(&state) % 100;  // 生成0到99之间的随机整数
    result[thread_id] = random_value;
}


void launch_sample_full(int* outputSRC1,
                 int* outputDST1,
                 const int* graphEdge,
                 const int* boundList,
                 const int* trainNode,
                 int n) {
    dim3 grid((n + 1023) / 1024);
    dim3 block(1024);
    sample_full_kernel<<<grid, block>>>(outputSRC1, outputDST1, graphEdge, boundList, trainNode, n);
}

void launch_sample_1hop(int* outputSRC1,
                        int* outputDST1, 
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int nodeNUM) {
    dim3 grid((nodeNUM + 1023) / 1024);
    dim3 block(1024);
    sample1Hop<<<grid, block>>>(
        outputSRC1,outputDST1,graphEdge,
        boundList,trainNode,sampleNUM1,
        nodeNUM);
}

void launch_sample_2hop(int* outputSRC1,
                        int* outputDST1,
                        int* outputSRC2,
                        int* outputDST2,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int sampleNUM2,
                        int nodeNUM) {
    dim3 grid((nodeNUM + 1023) / 1024);
    dim3 block(1024);
    sample2Hop<<<grid, block>>>(
        outputSRC1,outputDST1,outputSRC2,
        outputDST2,graphEdge,boundList,
        trainNode,sampleNUM1,sampleNUM2,nodeNUM);
}

void launch_sample_3hop(int* outputSRC1,int* outputDST1,
                        int* outputSRC2,int* outputDST2,
                        int* outputSRC3,int* outputDST3,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,int sampleNUM2,int sampleNUM3,
                        int nodeNUM) {
    dim3 grid((nodeNUM + 1023) / 1024);
    dim3 block(1024);
    sample3Hop<<<grid, block>>>(
        outputSRC1,outputDST1,outputSRC2,
        outputDST2,outputSRC3,outputDST3,
        graphEdge,boundList,trainNode,
        sampleNUM1,sampleNUM2,sampleNUM3,nodeNUM);
}