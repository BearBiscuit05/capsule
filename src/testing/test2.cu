#include <hipcub/hipcub.hpp>
#include <iostream>

int main() {
    const int num_elements = 8;
    int h_input[num_elements] = {2, 3, 1, 4, 2, 5, 6, 3};

    int *d_input, *d_output, *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // 在GPU上分配内存
    hipMalloc(&d_input, sizeof(int) * num_elements);
    hipMalloc(&d_output, sizeof(int) * num_elements);

    // 将输入数据拷贝到GPU
    hipMemcpy(d_input, h_input, sizeof(int) * num_elements, hipMemcpyHostToDevice);

    // 查询所需的临时存储空间大小
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_input, d_output, num_elements);

    // 分配临时存储空间
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // 执行并行前缀和操作
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_input, d_output, num_elements);

    // 将计算结果从GPU拷贝回主机
    int h_output[num_elements];
    hipMemcpy(h_output, d_output, sizeof(int) * num_elements, hipMemcpyDeviceToHost);

    // 打印计算结果
    std::cout << "Input array: ";
    for (int i = 0; i < num_elements; ++i) {
        std::cout << h_input[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Prefix sum array: ";
    for (int i = 0; i < num_elements; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // 释放内存
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_temp_storage);

    return 0;
}
