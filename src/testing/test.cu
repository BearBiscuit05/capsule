#include "hip/hip_runtime.h"
#include <stdio.h>    
#include <stdlib.h>   
#include <hip/hip_runtime.h>  
 
#define SIZE 8

__device__ void device_kernel(unsigned int *histo,int i) 
{
	atomicAdd(histo, i);
}


__global__ void histo_kernel(int size, unsigned int *histo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size)
	{
		device_kernel(histo, i);
	}
}

__global__ void histo(int size, unsigned int *histo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size)
	{
		device_kernel(histo, i);
	}
}

int main(void)
{
	int threadSum = 0;
 
	//分配内存并拷贝初始数据
	unsigned int *dev_histo;
 
	hipMalloc((void**)&dev_histo, sizeof(int));
	hipMemcpy(dev_histo, &threadSum, sizeof(int), hipMemcpyHostToDevice);
 
	// kernel launch - 2x the number of mps gave best timing  
	hipDeviceProp_t  prop;
	hipGetDeviceProperties(&prop, 0);
 
	int blocks = 2;
	//确保线程数足够
	histo_kernel << <blocks * 2, (SIZE + 2 * blocks - 1) / blocks / 2 >> > (SIZE, dev_histo);
	hipMemcpy(&threadSum, dev_histo, sizeof(int), hipMemcpyDeviceToHost);
	printf("Threads SUM：%d\n", threadSum);
	histo <<<blocks * 2, (SIZE + 2 * blocks - 1) / blocks / 2 >>> (SIZE, dev_histo);
	printf("Threads SUM：%d\n", threadSum);
	//数据拷贝回CPU内存
	hipMemcpy(&threadSum, dev_histo, sizeof(int), hipMemcpyDeviceToHost);
	printf("Threads SUM：%d\n", threadSum);
	hipFree(dev_histo);
	return 0;
}